#include "hip/hip_runtime.h"
#include "Renderer.h"
#include "cuda_gl_interop.h"
#include <iostream>
#include <time.h>



#ifdef DEBUG
#define HANDLE_ERROR(x){\
	hipError_t cudaStatus = (x); \
if (cudaStatus != hipSuccess){\
	fprintf(stdout, ": %s\n", hipGetErrorString(cudaStatus)); \
	system("Pause"); \
	exit(1); \
} \
}
#else
#define HANDLE_ERROR(x) x
#endif



void glfw_error_callback(int error, const char* description){
	fprintf(stderr, "Glfw Error %d: %s\n", error, description);
}

void newGLCheckError(const char *filename, const int line){
	GLenum err;
	char str[64];

	for (int i = 0; i < 5; i++){
		if ((err = glGetError()) != GL_NO_ERROR){
			switch (err){
			case GL_INVALID_ENUM:
				strcpy_s(str, "GL_INVALID_ENUM");
				break;
			case GL_INVALID_VALUE:
				strcpy_s(str, "GL_INVALID_VALUE");
				break;
			case GL_INVALID_OPERATION:
				strcpy_s(str, "GL_INVALID_OPERATION");
				break;
			case GL_STACK_OVERFLOW:
				strcpy_s(str, "GL_STACK_OVERFLOW");
				break;
			case GL_STACK_UNDERFLOW:
				strcpy_s(str, "GL_STACK_UNDERFLOW");
				break;
			case GL_OUT_OF_MEMORY:
				strcpy_s(str, "GL_OUT_OF_MEMORY");
				break;
				/*case GL_INVALID_FRAMEBUFFER_OPERATION:
				strcpy_s(str, "GL_INVALID_FRAMEBUFFER_OPERATION");
				break;*/
			default: break;
			}
			printf("GL error %ss %#x in file %s in line %d \n", str, err, filename, line);
		}
		else break;
	}
}


ImgSizePos getCorrectSize(int screenW, int screenH, int imgW, int imgH){
	float screenRatio = float(screenW) / float(screenH);
	float imageRatio = float(imgW) / float(imgH);
	ImgSizePos sizePos;
	if (imageRatio > screenRatio){
		sizePos.w = 1.0f;
		sizePos.h = screenRatio / imageRatio;
		sizePos.x = 0.0f;
		sizePos.y = 0.0f;
	}
	else{
		sizePos.w = imageRatio / screenRatio;
		sizePos.h = 1.0f;
		sizePos.x = 0.0f;
		sizePos.y = 0.0f;
	}
	return sizePos;
}
inline float clamp(float x){ return x < 0.0f ? 0.0f : x > 1.0f ? 1.0f : x; }
inline int toInt(float x){ return int(clamp(x) * 255 + .5); }


void Renderer::postProcess(){
	glBindFramebuffer(GL_FRAMEBUFFER, fbos[FBO::FBO2]);
	glViewport(0, 0, width, height);
	glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
	glClear(GL_COLOR_BUFFER_BIT);
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, textures[TEXTURE::BUFFER1]);

	glBindVertexArray(vaos[VAO::ScreenGeomVao]);
	programs[ShaderEnum::DrawBuffer].bind();
	glUniform1f(programs[ShaderEnum::DrawBuffer].getUniformLocation("exposure"), glm::pow(2.0f, exposure));
	glUniform1f(programs[ShaderEnum::DrawBuffer].getUniformLocation("rays"), float(rays));
	glUniform1i(programs[ShaderEnum::DrawBuffer].getUniformLocation("buffer"), 0);
	glDrawArrays(GL_TRIANGLES, 0, 6);

	glEnable(GL_LINE_SMOOTH);
	programs[ShaderEnum::DrawShape].bind();
	glUniform1f(programs[ShaderEnum::DrawShape].getUniformLocation("aspectRatio"), float(width) / height);
	for (int i = 0; i < objects.size(); i++){
		glUniformMatrix3fv(programs[ShaderEnum::DrawShape].getUniformLocation("matrix"), 1, GL_FALSE, glm::value_ptr(objects[i].t.inv));
		if (objects[i].objectType == 0){
			glBindVertexArray(vaos[VAO::LineShapeVao]);
			glDrawArrays(GL_LINES, 0, 2);
		}
		else if (objects[i].objectType == 1){
			glBindVertexArray(vaos[VAO::CircleShapeVao]);
			glDrawArrays(GL_LINE_LOOP, 0, CIRCLE_TESSELATION);
		}
	}
}
void Renderer::saveFrame(std::string filename){
	postProcess();
	std::vector<float3> data(width * height);
	glBindTexture(GL_TEXTURE_2D, textures[TEXTURE::OFFSCREEN]);
	glGetTexImage(GL_TEXTURE_2D, 0, GL_RGB, GL_FLOAT, &data[0]);
	FreeImage_Initialise();
	FIBITMAP *bitmap = FreeImage_Allocate(width, height, 24);
	for (int j = 0; j < height; j++)
	for (int i = 0; i < width; i++){
		RGBQUAD rgb;
		float3 frgb = data[i + j*width];
		rgb.rgbRed = (BYTE)toInt(frgb.x);
		rgb.rgbGreen = (BYTE)toInt(frgb.y);
		rgb.rgbBlue = (BYTE)toInt(frgb.z);
		FreeImage_SetPixelColor(bitmap, i, j, &rgb);
	}
	filename += ".png";
	FreeImage_Save(FREE_IMAGE_FORMAT::FIF_PNG, bitmap, filename.c_str());
	FreeImage_Unload(bitmap);
	FreeImage_DeInitialise();
}

void Renderer::saveLineFrames(std::string filename, int framerate, int frames){
	initLineBuffer();
	setupFrame();
	for (int i = 0; i < frames; i++){
		printf("Frame : %d\n", i);
		HANDLE_ERROR(hipMemset(dev_rayInfo, 0x00, rayBatchSize * sizeof(RayInfo)));
		HANDLE_ERROR(hipMemset(dev_rays, 0x00, sizeof(int)));
		float _time = float(i) / framerate;
		renderLineFrame(_time);
		saveFrame(filename + std::to_string(i + 1));
	}
	HANDLE_ERROR(hipGraphicsUnregisterResource(resourceVBO));
	hipFree(dev_rayInfo);
	hipFree(dev_states);
}
//render pulse animation
void Renderer::savePulseFrames(std::string filename, int framerate, int frames){
	initLineBuffer();
	setupFrame();
	for (int i = 0; i < frames; i++){
		printf("Frame : %d\n", i);
		HANDLE_ERROR(hipMemset(dev_rayInfo, 0x00, rayBatchSize * sizeof(RayInfo)));
		HANDLE_ERROR(hipMemset(dev_rays, 0x00, sizeof(int)));
		float _time = float(i) / framerate;
		renderPulseFrame(_time, dt);
		saveFrame(filename + std::to_string(i + 1));
	}
	HANDLE_ERROR(hipGraphicsUnregisterResource(resourceVBO));
	hipFree(dev_rayInfo);
	hipFree(dev_states);
}
/*
single frame rendering
*/
void Renderer::renderLineFrame(float time){
	int seed = rand();
	dim3 block(64, 1, 1);
	dim3 grid((int)ceil(float(rayBatchSize) / block.x), 1, 1);
	curandInitKernel << <grid, block >> >(dev_states, seed, rayBatchSize);
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipGetLastError());

	glBindFramebuffer(GL_FRAMEBUFFER, fbos[FBO::FBO1]);
	glViewport(0, 0, width, height);
	glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
	glClear(GL_COLOR_BUFFER_BIT);
	for (int i = 0; i < rayBatchCount; i++){
		//run cuda 
		size_t num_bytes = 0;
		HANDLE_ERROR(hipGraphicsMapResources(1, &resourceVBO, 0));
		HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void **)&dev_vbo,
			&num_bytes, resourceVBO));
		kernelLine << < grid, block >> >(dev_rayInfo, dev_vbo, cudaScene, dev_states, dev_rays, rayBatchSize, time);
		HANDLE_ERROR(hipDeviceSynchronize());
		HANDLE_ERROR(hipGetLastError());
		if (i != rayBatchCount - 1){
			printf("\rProgress %d    ", i + 1);
			Sleep(sleepAmount);
		}
		HANDLE_ERROR(hipGraphicsUnmapResources(1, &resourceVBO, 0));

		glLineWidth((float)size);
		glDisable(GL_DEPTH_TEST);
		glDisable(GL_CULL_FACE);
		//glEnable(GL_LINE_SMOOTH);
		glEnable(GL_BLEND);
		glBlendFunc(GL_ONE, GL_ONE);
		glBlendEquation(GL_FUNC_ADD);
		programs[1].bind();
		glUniform1f(programs[1].getUniformLocation("aspectRatio"), float(width) / height);
		glUniform2f(programs[1].getUniformLocation("screenSize"), (float)width, (float)height);
		glBindVertexArray(vaos[VAO::LineVao]);
		glDrawArrays(GL_LINES, 0, rayBatchSize * 2);
	}
	HANDLE_ERROR(hipMemcpy((void*)&rays, dev_rays, sizeof(int), hipMemcpyDeviceToHost));
	printf("\n");
}
void Renderer::renderPulseFrame(float time, float dt){
	int seed = rand();
	dim3 block(64, 1, 1);
	dim3 grid((int)ceil(float(rayBatchSize) / block.x), 1, 1);
	curandInitKernel << <grid, block >> >(dev_states, seed, rayBatchSize);

	glBindFramebuffer(GL_FRAMEBUFFER, fbos[FBO::FBO1]);
	glViewport(0, 0, width, height);
	glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
	glClear(GL_COLOR_BUFFER_BIT);

	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipGetLastError());
	for (int i = 0; i < rayBatchCount; i++){
		//run cuda 
		size_t num_bytes = 0;
		HANDLE_ERROR(hipGraphicsMapResources(1, &resourceVBO, 0));
		HANDLE_ERROR(hipGraphicsResourceGetMappedPointer((void **)&dev_vbo,
			&num_bytes, resourceVBO));
		kernelLinePulse << < grid, block >> >(dev_rayInfo, dev_vbo, cudaScene, dev_states, dev_rays, rayBatchSize, time, dt);
		HANDLE_ERROR(hipDeviceSynchronize());
		HANDLE_ERROR(hipGetLastError());
		if (i != rayBatchCount - 1){
			printf("\rProgress %d    ", i + 1);
			Sleep(sleepAmount);
		}
		HANDLE_ERROR(hipGraphicsUnmapResources(1, &resourceVBO, 0));

		glLineWidth((float)size);
		glDisable(GL_DEPTH_TEST);
		glDisable(GL_CULL_FACE);
		//glEnable(GL_LINE_SMOOTH);
		glEnable(GL_BLEND);
		glBlendFunc(GL_ONE, GL_ONE);
		glBlendEquation(GL_FUNC_ADD);
		programs[1].bind();
		glUniform1f(programs[1].getUniformLocation("aspectRatio"), float(width) / height);
		glUniform2f(programs[1].getUniformLocation("screenSize"), (float)width, (float)height);
		glBindVertexArray(vaos[VAO::LineVao]);
		glDrawArrays(GL_LINES, 0, rayBatchSize * 2);
	}
	HANDLE_ERROR(hipMemcpy((void*)&rays, dev_rays, sizeof(int), hipMemcpyDeviceToHost));
	printf("\n");
}
//render full lines
void Renderer::renderLines(float time){
	initLineBuffer();
	setupFrame();
	renderLineFrame(time);
	HANDLE_ERROR(hipGraphicsUnregisterResource(resourceVBO));
	hipFree(dev_rayInfo);
	hipFree(dev_states);
}

//render pulses
void Renderer::renderPulses(float time, float dt){
	initLineBuffer();
	setupFrame();
	renderPulseFrame(time, dt);
	HANDLE_ERROR(hipGraphicsUnregisterResource(resourceVBO));
	hipFree(dev_rayInfo);
	hipFree(dev_states);
}


Renderer::Renderer(){
	cudaScene.lights = NULL;
	cudaScene.objects = NULL;
	cudaScene.materials = NULL;
}
Renderer::~Renderer(){
	glDeleteTextures(TEXTURECOUNT, textures);
	glDeleteFramebuffers(FBOCOUNT, fbos);

	hipFree(dev_rays);
	hipFree(cudaScene.lights);
	hipFree(cudaScene.objects);
	hipFree(cudaScene.materials);
}
void Renderer::start(){
	glfwSetErrorCallback(glfw_error_callback);
	if (!glfwInit())
		return;
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	GLFWwindow* window = glfwCreateWindow(1024, 1024, "Cuda forward tracing 2d", NULL, NULL);
	if (window == NULL){
		glfwTerminate();
		return;
	}
	glfwMakeContextCurrent(window);
	glewExperimental = GL_TRUE;
	GLenum glew_status = glewInit();
	if (glew_status != GLEW_OK){
		fprintf(stdout, "Error: %s\n", glewGetErrorString(glew_status));
		return;
	}
	glGetError();
	glfwSwapInterval(1);

	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO();

	const char* glsl_version = "#version 330";
	ImGui::StyleColorsDark();
	ImGui_ImplGlfw_InitForOpenGL(window, true);
	ImGui_ImplOpenGL3_Init(glsl_version);

	init();
	initScene();
	exposure = 1.0f;
	rayBatchSize = 2048;
	rayBatchCount = 100;
	sleepAmount = 20;
	size = 1;
	timeFrame = 0.0;
	frameRate = 24;
	frames = 240;
	w = 1280;
	h = 720;
	dt = 100.0f;
	srand(time(NULL));

	char buf[64];
	memset(buf, 0x00, sizeof(char)* 64);
	while (!glfwWindowShouldClose(window)){
		glfwPollEvents();
		ImGui_ImplOpenGL3_NewFrame();
		ImGui_ImplGlfw_NewFrame();
		ImGui::NewFrame();
		{
			ImGui::Begin("main");
			ImGui::SliderFloat("exposure", &exposure, -100.0f, 100.0f);
			ImGui::DragInt("width", &w, 1.0f, 200, 1920);
			ImGui::DragInt("height", &h, 1.0f, 200, 1080);
			ImGui::DragInt("ray batch size", &rayBatchSize, 1.0f, 64, 8192);
			ImGui::DragInt("ray batch count", &rayBatchCount, 1.0f, 1, 10000);
			ImGui::DragFloat("time frame", &timeFrame, 0.1f, 0.0f, 1000.0f);
			ImGui::DragFloat("dt", &dt, 0.1f, 0.0f, 1000.0f);
			if (ImGui::Button("load scene")){
				loadScene();
			}
			if (ImGui::Button("render frame")){
				width = w;
				height = h;
				renderLines(timeFrame);
			}
			if (ImGui::Button("render frame pulse")){
				width = w;
				height = h;
				renderPulses(timeFrame, dt);
			}
			ImGui::End();
			//render fbo
			ImGui::Begin("anim");
			ImGui::DragInt("framerate", &frameRate, 1.0f, 1, 60);
			ImGui::DragInt("frames", &frames, 1.0f, 1, 10000);
			ImGui::InputText("filename", buf, IM_ARRAYSIZE(buf));
			if (ImGui::Button("render full line anim")){
				width = w;
				height = h;
				saveLineFrames(std::string(buf), frameRate, frames);
			}
			if (ImGui::Button("render pulse anim")){
				width = w;
				height = h;
				savePulseFrames(std::string(buf), frameRate, frames);
			}
			ImGui::End();
		}
		ImGui::Render();
		glfwMakeContextCurrent(window);

		int screenWidth, screenHeight;
		glfwGetFramebufferSize(window, &screenWidth, &screenHeight);

		postProcess();

		glBindFramebuffer(GL_FRAMEBUFFER, 0);
		glViewport(0, 0, screenWidth, screenHeight);
		glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
		glClear(GL_COLOR_BUFFER_BIT);
		glActiveTexture(GL_TEXTURE0);
		glBindTexture(GL_TEXTURE_2D, textures[TEXTURE::OFFSCREEN]);
		programs[ShaderEnum::DrawToScreen].bind();
		ImgSizePos imgSizePos = getCorrectSize(screenWidth, screenHeight, width, height);
		glm::mat3 mvp = glm::mat3(1.0f);
		mvp = glm::translate(mvp, glm::vec2(imgSizePos.x, imgSizePos.y));
		mvp = glm::scale(mvp, glm::vec2(imgSizePos.w, imgSizePos.h));
		//mvp = glm::inverse(mvp);
		glBindVertexArray(vaos[VAO::ScreenGeomVao]);
		glUniform1i(programs[ShaderEnum::DrawToScreen].getUniformLocation("buffer"), 0);
		glUniformMatrix3fv(programs[ShaderEnum::DrawToScreen].getUniformLocation("mvp"), 1, GL_FALSE, glm::value_ptr(mvp));
		glDrawArrays(GL_TRIANGLES, 0, 6);
		programs[2].unbind();
		//!!
		ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

		glfwMakeContextCurrent(window);
		glfwSwapBuffers(window);
	}
	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();
	glfwDestroyWindow(window);
	glfwTerminate();
}

void Renderer::init(){
	const char* shaderNames[] = {
		"Shaders/drawBuffer.vert", "Shaders/drawBuffer.frag",
		"Shaders/lineShader.vert", "Shaders/lineShader.frag",
		"Shaders/drawScreen.vert", "Shaders/drawScreen.frag",
		"Shaders/drawShape.vert", "Shaders/drawShape.frag"
	};
	for (int i = 0; i < SHADERCOUNT; i++){
		if (!programs[i].loadShaderProgram(shaderNames[i * 2], shaderNames[i * 2 + 1])){
			printf("Error loading shader file.\n");
			system("pause");
			exit(1);
		}
	}
	//INIT FBO AND TEXURES
	glGenFramebuffers(FBOCOUNT, fbos);
	glGenTextures(TEXTURE::TEXTURECOUNT, textures);

	float white[3] = { 1.0f, 1.0f, 1.0f };
	width = 1;
	height = 1;
	glBindTexture(GL_TEXTURE_2D, textures[TEXTURE::BUFFER1]);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB32F, width, height, 0, GL_RGB, GL_FLOAT, white);

	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);


	//create vaos and vbos
	glGenVertexArrays(VAOCOUNT, vaos);
	glGenBuffers(VBOCOUNT, vbos);

	glBindVertexArray(vaos[LineVao]);
	glBindBuffer(GL_ARRAY_BUFFER, vbos[LightInfoVBO]);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(VBOInfo), 0);
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(VBOInfo), (void *)(4 * sizeof(float)));
	glEnableVertexAttribArray(2);
	glVertexAttribPointer(2, 1, GL_FLOAT, GL_FALSE, sizeof(VBOInfo), (void *)(7 * sizeof(float)));

	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);


	programs[LineShader].bindAttribLocation(0, "in_pos");
	programs[LineShader].bindAttribLocation(1, "in_color");
	programs[LineShader].bindAttribLocation(2, "in_angle");
	programs[LineShader].link();
	/*
	programs[PointShader].bindAttribLocation(0, "in_pos");
	programs[PointShader].bindAttribLocation(1, "in_color");
	programs[PointShader].bindAttribLocation(2, "in_angle");
	programs[PointShader].link();*/

	float fullscreenRect[] = { -1.0, -1.0
		, -1.0, 1.0
		, 1.0, 1.0
		, -1.0, -1.0
		, 1.0, 1.0
		, 1.0, -1.0
	};
	glBindVertexArray(vaos[ScreenGeomVao]);
	glBindBuffer(GL_ARRAY_BUFFER, vbos[VBO::ScreenGeomVBO]);
	glBufferData(GL_ARRAY_BUFFER, sizeof(float)* 12, fullscreenRect, GL_STATIC_DRAW);

	programs[0].bindAttribLocation(0, "pos");
	programs[0].link();
	programs[2].bindAttribLocation(0, "pos");
	programs[2].link();
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 0, 0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);

	programs[DrawShape].bindAttribLocation(0, "in_pos");
	programs[DrawShape].link();

	glBindVertexArray(vaos[VAO::LineShapeVao]);
	glBindBuffer(GL_ARRAY_BUFFER, vbos[VBO::LineVBO]);
	float line[] = { 0.0f, 0.0f, 1.0f, 0.0f };
	glBufferData(GL_ARRAY_BUFFER, sizeof(float)* 4, line, GL_STATIC_DRAW);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 0, 0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);

	glBindVertexArray(vaos[VAO::CircleShapeVao]);
	glBindBuffer(GL_ARRAY_BUFFER, vbos[VBO::CircleVBO]);
	std::vector<float> circleVectors(CIRCLE_TESSELATION * 2);
	for (int i = 0; i < CIRCLE_TESSELATION; i++){
		circleVectors[i * 2] = glm::cos(i*glm::two_pi<float>() / CIRCLE_TESSELATION);
		circleVectors[i * 2 + 1] = glm::sin(i*glm::two_pi<float>() / CIRCLE_TESSELATION);
	}
	glBufferData(GL_ARRAY_BUFFER, sizeof(float)* 2 * CIRCLE_TESSELATION, &circleVectors[0], GL_STATIC_DRAW);
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 0, 0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);

	CheckGl();

	HANDLE_ERROR(hipMalloc((void**)&dev_rays, sizeof(int)));
}
void Renderer::initScene(){
	SceneDescription scene;
	Light light;
	light.lightType = _POINT;
	light.color = glm::vec3(1.0,1.,0.);
	light.angle = 0.0;
	light.t = createTransform(glm::vec2(0.0f, 0.0f), glm::radians(0.0f), glm::vec2(0.01f));
	scene.lights.push_back(light);

	initCudaScene(scene);
}

void Renderer::initCudaScene(const SceneDescription& scene){
	objects.clear();
	//could be null pointers
	hipFree(cudaScene.lights);
	hipFree(cudaScene.objects);
	hipFree(cudaScene.materials);
	for (auto iter = scene.objects.begin(); iter != scene.objects.end(); ++iter){
		objects.push_back(*iter);
	}
	cudaScene.lightCount = scene.lights.size();
	cudaScene.matCount = scene.materials.size();
	cudaScene.objCount = scene.objects.size();
	cudaScene.minDepth = 3;
	cudaScene.maxDepth = 8;
	if (cudaScene.matCount>0)
		HANDLE_ERROR(hipMalloc((void**)&(cudaScene.materials), cudaScene.matCount * sizeof(Material)));
	if (cudaScene.objCount>0)
		HANDLE_ERROR(hipMalloc((void**)&(cudaScene.objects), cudaScene.objCount * sizeof(Object)));
	if (cudaScene.lightCount>0)
		HANDLE_ERROR(hipMalloc((void**)&(cudaScene.lights), cudaScene.lightCount * sizeof(Light)));

	if (cudaScene.matCount>0)
		HANDLE_ERROR(hipMemcpy(cudaScene.materials, scene.materials.data(), cudaScene.matCount * sizeof(Material), hipMemcpyHostToDevice));
	if (cudaScene.objCount>0)
		HANDLE_ERROR(hipMemcpy(cudaScene.objects, scene.objects.data(), cudaScene.objCount * sizeof(Object), hipMemcpyHostToDevice));

	if (cudaScene.lightCount>0)
		HANDLE_ERROR(hipMemcpy(cudaScene.lights, scene.lights.data(), cudaScene.lightCount * sizeof(Light), hipMemcpyHostToDevice));
}
void Renderer::initLineBuffer(){
	glBindBuffer(GL_ARRAY_BUFFER, vbos[VBO::LightInfoVBO]);
	glBufferData(GL_ARRAY_BUFFER, sizeof(VBOInfo)* rayBatchSize * 2, NULL, GL_DYNAMIC_DRAW);

	resourceVBO = NULL;
	HANDLE_ERROR(hipGraphicsGLRegisterBuffer(&resourceVBO, vbos[VBO::LightInfoVBO], cudaGraphicsMapFlagsWriteDiscard));

}


void Renderer::setupFrame(){
	glBindTexture(GL_TEXTURE_2D, textures[TEXTURE::BUFFER1]);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA, GL_FLOAT, NULL);

	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

	glBindTexture(GL_TEXTURE_2D, textures[TEXTURE::OFFSCREEN]);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB32F, width, height, 0, GL_RGB, GL_FLOAT, NULL);

	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

	glBindFramebuffer(GL_FRAMEBUFFER, fbos[FBO::FBO1]);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, textures[TEXTURE::BUFFER1], 0);
	if (glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE){
		fprintf(stdout, "fbo1 attachement failed\n"); \
			system("pause"); \
			exit(1);
	}

	glBindFramebuffer(GL_FRAMEBUFFER, fbos[FBO::FBO2]);
	glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, textures[TEXTURE::OFFSCREEN], 0);
	if (glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE){
		fprintf(stdout, "fbo2 attachement failed\n"); \
			system("pause"); \
			exit(1);
	}

	//setup cuda

	HANDLE_ERROR(hipMalloc((void**)&dev_rayInfo, rayBatchSize * sizeof(RayInfo)));
	HANDLE_ERROR(hipMalloc((void**)&dev_states, rayBatchSize*sizeof(hiprandState_t)));
	HANDLE_ERROR(hipMemset(dev_rayInfo, 0x00, rayBatchSize * sizeof(RayInfo)));
	HANDLE_ERROR(hipMemset(dev_rays, 0x00, sizeof(int)));
}
void Renderer::loadScene(){
	throw "Not implemented";
}
