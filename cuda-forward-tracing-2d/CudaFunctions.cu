#include "hip/hip_runtime.h"
#include "CudaFunctions.cuh"




#undef max
#undef min





__host__ Transform createTransform(glm::vec2 t, float r, glm::vec2 s){
	glm::mat3 mat = glm::mat3(1.0f);
	mat = glm::translate(mat, t);
	mat = glm::rotate(mat, r);
	mat = glm::scale(mat, s);
	glm::mat3 invMat = inverse(mat);
	return Transform{ invMat, mat };
}

__device__ float rand1f(hiprandState_t* state){
	return hiprand_uniform(state);
}

__device__ float toWaveIntensity(glm::vec3 rgb, float wavelength){
	float number = (wavelength - 380.0f) / (730.0f - 380.0f);
	number = glm::max(glm::min(1.0f, number), 0.0f);
	int bin = glm::floor(number * 35.0f);
	int next = glm::max(glm::min(35, bin + 1), 0);
	return glm::dot(glm::mix(glm::vec3(rho_R[bin], rho_G[bin], rho_B[bin]), glm::vec3(rho_R[next], rho_G[next], rho_B[next]), glm::fract(number*35.0f)), rgb);
}
__device__ float xFit_1931(float wave){
	float t1 = (wave - 442.0)*((wave < 442.0) ? 0.0624 : 0.0374);
	float t2 = (wave - 599.8)*((wave < 599.8) ? 0.0264 : 0.0323);
	float t3 = (wave - 501.1)*((wave < 501.1) ? 0.0490 : 0.0382);
	return 0.362*glm::exp(-0.5*t1*t1) + 1.056*glm::exp(-0.5*t2*t2)
		- 0.065*glm::exp(-0.5*t3*t3);
}
__device__ float yFit_1931(float wave){
	float t1 = (wave - 568.8)*((wave < 568.8) ? 0.0213 : 0.0247);
	float t2 = (wave - 530.9)*((wave < 530.9) ? 0.0613 : 0.0322);
	return 0.821*glm::exp(-0.5*t1*t1) + 0.286*glm::exp(-0.5*t2*t2);
}
__device__ float zFit_1931(float wave){
	float t1 = (wave - 437.0)*((wave < 437.0) ? 0.0845 : 0.0278);
	float t2 = (wave - 459.0)*((wave < 459.0) ? 0.0385 : 0.0725);
	return 1.217*glm::exp(-0.5*t1*t1) + 0.681*glm::exp(-0.5*t2*t2);
}
__host__ __device__ glm::vec3 XYZToRGB(glm::vec3 xyz){
	const glm::mat3 XYZ_to_RGB(2.3706743f, -0.9000405f, -0.4706338f,
		-0.5138850f, 1.4253036f, 0.0885814f,
		0.0052982f, -0.0146949f, 1.0093968f);
	return xyz*XYZ_to_RGB;
}
//circle at 0,0 with radius 1
__device__ float circle(const Ray2D& ray, glm::vec2& normal){
	float a = glm::dot(ray.rd, ray.rd);
	float b = glm::dot(ray.rd, ray.ro);
	float c = glm::dot(ray.ro, ray.ro) - 1.0f;
	float d = b * b - c * a;
	if (d >= 0.0){
		d = sqrt(d);
		float t = (-b - d) / a;
		if (t < 0.0){
			t = (-b + d) / a;
		}
		normal = glm::normalize(ray.ro + ray.rd*t);
		return t;
	}
	return -1.0f;
}
//line with points 0,0 and 1,0
__device__ float line(const Ray2D& ray, glm::vec2& normal){
	float t = (-ray.ro.y) / ray.rd.y;
	if (t >= 0){
		glm::vec2 point = ray.ro + ray.rd*t;
		if (glm::abs(point.x - 0.5f) <= 0.5f){
			normal.x = 0.0f;
			normal.y = 1.0f;
			return t;
		}
	}
	return -1.0f;
}
__device__ float intersectShape(int shapeType, const Ray2D& ray, glm::vec2& normal){
	switch (shapeType){
		case 0:
			return line(ray, normal);
		case 1:
			return circle(ray, normal);
	}
	return -1.0f;
}
__device__ float intersectObject(const Ray2D& ray, const Object& object, glm::vec2& normal){
	Ray2D tRay;
	tRay = worldToLocal(object.t, ray);
	float t = intersectShape(object.objectType, tRay, normal);
	normal = glm::normalize(localToWorldN(object.t, normal));
	return t;
}
__device__ float intersectLight(const Ray2D& ray, Light& light){
	switch (light.lightType){
	case _POINT:
		break;
	case POINTDIRECTIONAL:
		break;
	case POINTANGULAR:
		break;
	case CIRCLE:{
				   float a = glm::dot(ray.rd, ray.rd);
				   float b = glm::dot(ray.rd, ray.ro);
				   float c = glm::dot(ray.ro, ray.ro) - 1.0f;
				   float d = b * b - c * a;
				   if (d >= 0.0){
					   d = sqrt(d);
					   float t = (-b - d) / a;
					   if (t < 0.0){
						   t = (-b + d) / a;
					   }
					   return t;
				   }
				   break;
	}
	case CIRCLEANGULAR:{
						  float a = glm::dot(ray.rd, ray.rd);
						  float b = glm::dot(ray.rd, ray.ro);
						  float c = glm::dot(ray.ro, ray.ro) - 1.0f;
						  float d = b * b - c * a;
						  if (d >= 0.0){
							  float ax = -glm::cos(light.angle);
							  d = sqrt(d);
							  float t = (-b - d) / a;
							  if (t < 0.0){
								  t = (-b + d) / a;
							  }
							  else{
								  glm::vec2 p = ray.ro + ray.rd*t;
								  if (p.x < ax){
									  t = (-b + d) / a;
									  p = ray.ro + ray.rd*t;
									  if (p.x < ax)
										  return -1.0f;
								  }
							  }
							  return t;
						  }
						  break;
	}
	case LINE:
	case LINEDIRECTIONAL:{
							float t = (-ray.ro.x) / ray.rd.x;
							if (t >= 0){
								glm::vec2 point = ray.ro + ray.rd*t;
								if (glm::abs(point.y) <= 0.5f){
									return t;
								}
							}
	}
	}
	return -1.0f;
}
__device__ bool findIntersection(const Ray2D&ray, Object* objects, int objCount, Light* lights, int lightCount, HitInfo& info){
	float t0 = 10e8;
	int id = -1;
	for (int i = 0; i < objCount; i++){
		glm::vec2 normal;
		float t = intersectObject(ray, objects[i], normal);
		if (t <= t0 && t >= 0.0){
			t0 = t;
			info.normal = normal;
			info.material = objects[i].materialType;
			id = i;
		}
	}
	bool lightInter = false;
	for (int i = 0; i < lightCount; i++){
		Ray2D tRay;
		tRay = worldToLocal(lights[i].t, ray);
		float t = intersectLight(tRay, lights[i]);
		if (t <= t0 && t >= 0.0){
			lightInter = true;
			info.material = -1;
			t0 = t;
		}
	}
	info.point = ray.ro + ray.rd*t0;
	if (lightInter)
		return true;
	if (id == -1)
		return false;
	switch (objects[id].normalType){
		case NormalType::B:
			info.normal = -info.normal;
			break;
		case NormalType::BOTH:
			info.normal = info.normal * glm::sign(-glm::dot(info.normal, ray.rd));
			break;
	}
	return true;
}

__device__ Ray2D localToWorld(const Transform& t, const Ray2D&ray){
	Ray2D result;
	result.ro = glm::vec2(t.inv*glm::vec3(ray.ro, 1.0f));
	result.rd = glm::vec2(t.inv*glm::vec3(ray.rd, 0.0f));
	return result;
}
__device__ Ray2D worldToLocal(const Transform& t, const Ray2D&ray){
	Ray2D result;
	result.ro = glm::vec2(t.t*glm::vec3(ray.ro, 1.0f));
	result.rd = glm::vec2(t.t*glm::vec3(ray.rd, 0.0f));
	return result;
}
__device__ glm::vec2 localToWorldN(const Transform& t, glm::vec2& normal){
	return glm::vec2(glm::vec3(normal, 0.0f)*t.t);
}
__device__ glm::vec2 getUniformCircle(hiprandState_t* state){
	float angle = rand1f(state)*glm::two_pi<float>();
	return glm::vec2(glm::cos(angle), glm::sin(angle));
}
__device__ glm::vec2 getUniformHemiCircle(glm::vec2 dir, hiprandState_t* state){
	float angle = rand1f(state)*glm::pi<float>();
	glm::vec2 tang(-dir.y, dir.x);
	return glm::cos(angle)*tang + glm::sin(angle)*dir;
}
__device__ glm::vec2 getCosDistribution(glm::vec2 dir, hiprandState_t* state){
	glm::vec2 normal(-dir.y, dir.x);
	float x = rand1f(state)*2.0f - 1.0f;
	return normal * x + dir * glm::sqrt(1.0f - x * x);
}
__device__ Ray2D sampleRay(const Light* light, float* pdf, hiprandState_t* state){
	Ray2D ray;
	switch (light->lightType){
	case _POINT:{
				   ray.ro = glm::vec2(0.0f, 0.0f);
				   ray.rd = getUniformCircle(state);
				   *pdf = glm::one_over_two_pi<float>();
				   break;
	}
	case POINTDIRECTIONAL:{
							 ray.ro = glm::vec2(0.0f, 0.0f);
							 ray.rd = glm::vec2(1.0f, 0.0f);
							 *pdf = 1.0f;
							 //return one possible ray
							 break;
	}
	case POINTANGULAR:{
						 ray.ro = glm::vec2(0.0f, 0.0f);
						 float angle = (rand1f(state) - 0.5f)*light->angle;
						 ray.rd = glm::vec2(glm::cos(angle), glm::sin(angle));
						 ray.ro += ray.rd*0.001f;
						 *pdf = 1.0f / light->angle;
						 break;
	}
	case CIRCLE:{
				   ray.ro = getUniformCircle(state);
				   ray.rd = getUniformCircle(state);
				   *pdf = glm::one_over_two_pi<float>()*glm::one_over_two_pi<float>();
				   break;
	}
	case CIRCLEANGULAR:{
						  float angle = (rand1f(state) - 0.5f)*light->angle;
						  ray.ro = glm::vec2(glm::cos(angle), glm::sin(angle));
						  ray.rd = getUniformCircle(state);
						  *pdf = glm::one_over_two_pi<float>() / light->angle;
						  break;
	}
	case LINE:{
				 ray.ro = glm::vec2(0.0, rand1f(state) - 0.5f);
				 ray.rd = getUniformCircle(state);
				 *pdf = glm::one_over_two_pi<float>();//unit length line
				 break;
	}
	case LINEDIRECTIONAL:{
							ray.ro = glm::vec2(0.0, rand1f(state) - 0.5f);
							ray.rd = glm::vec2(1.0f, 0.0f);
							*pdf = 1.0f;//unit length line
							break;
	}
	}
	ray = localToWorld(light->t, ray);
	ray.rd = glm::normalize(ray.rd);
	return ray;
}
__device__ void swap(float& a, float& b){
	float t = a;
	a = b;
	b = t;
}
__device__ float frDielectric(float cosThetaI, float etaI, float etaT){
	cosThetaI = glm::clamp(cosThetaI, -1.0f, 1.0f);
	bool outside = cosThetaI > 0.0f;
	if (!outside){
		swap(etaI, etaT);
		cosThetaI = glm::abs(cosThetaI);
	}
	float sinThetaI = glm::sqrt(glm::max(0.0f, 1.0f - cosThetaI * cosThetaI));
	float sinThetaT = etaI / etaT * sinThetaI;
	float cosThetaT = glm::sqrt(glm::max(0.0f, 1.0f - sinThetaT * sinThetaT));
	float rparl = (etaT*cosThetaI - etaI * cosThetaT) /
		(etaT*cosThetaI + etaI * cosThetaT);
	float rperp = ((etaI * cosThetaI) - (etaT * cosThetaT)) /
		((etaI * cosThetaI) + (etaT * cosThetaT));
	return (rparl*rparl + rperp * rperp)*0.5f;
}
__device__ float getRefractionIndex(float wave, glm::vec3 b, glm::vec3 c){
	float waveSqr = wave * wave;
	glm::vec3 num = b * waveSqr;
	glm::vec3 denum = waveSqr - c;
	num = num / denum;
	return sqrt(1.0 + glm::dot(num, glm::vec3(1.0)));
}
__device__ glm::vec2 traceRect(const Ray2D& ray, glm::vec2 min, glm::vec2 max){
	float tmin, tmax, tymin, tymax;
	glm::vec2 invDir = 1.0f / ray.rd;
	tmin = ((ray.rd.x >= 0.0 ? min.x : max.x) - ray.ro.x) * invDir.x;
	tmax = ((ray.rd.x >= 0.0 ? max.x : min.x) - ray.ro.x) * invDir.x;
	tymin = ((ray.rd.y >= 0.0 ? min.y : max.y) - ray.ro.y) * invDir.y;
	tymax = ((ray.rd.y >= 0.0 ? max.y : min.y) - ray.ro.y) * invDir.y;
	if ((tmin > tymax) || (tymin > tmax))
		return glm::vec2(-1.0f, -1.0f);
	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;
	return glm::vec2(tmin, tmax);
}

__device__ Ray2D genRay(Light* lights, int lightCount, RayInfo &rayInfo, float& currentTime, hiprandState_t* state){
	int lightIndex = floor(rand1f(state)*lightCount);
	Light* light = lights+lightIndex;
	float pdf = 1.0f;
	Ray2D ray = sampleRay(light, &pdf, state);
	pdf /= lightCount;
	ray.ro += ray.rd*0.001f;

	rayInfo.ro = ray.ro;
	rayInfo.rd = ray.rd;
	rayInfo.time = 0;
	rayInfo.speed = 1.0f;
	currentTime = 0;
	float wavelength = glm::mix(spectrumLow, spectrumHigh, rand1f(state));
	float intensity = toWaveIntensity(light->color, wavelength)*pdf;
	rayInfo.wavelength = wavelength;
	rayInfo.color = intensity*glm::max(XYZToRGB(glm::vec3(xFit_1931(wavelength), yFit_1931(wavelength), zFit_1931(wavelength))), 0.0f);//color of one ray = power*pdf
	return ray;
}

__device__ void shade(Ray2D& ray, const HitInfo& hitInfo, Material * materials,RayInfo& rayInfo,hiprandState_t* state){
	glm::vec2 nr;
	Material* material = materials+hitInfo.material;
	glm::vec3 lr;
	float dotn = glm::dot(ray.rd, hitInfo.normal);
	rayInfo.depth++;
	glm::vec3 f(1.0f);
	float pdf = 1.0f;
	switch (material->type){
		case DIFFUSEREFL:{
			if (dotn >= 0.0f){
				rayInfo.depth = 0;
				break;
			}
			nr = getCosDistribution(hitInfo.normal, state);
			float dt = glm::dot(nr, hitInfo.normal);
			f = material->r*0.5f;
			pdf = abs(dt)*0.5f;
			break;
		}
		//sample random direction in hemicircle
		case DIFFUSETRANSM:{
			nr = getCosDistribution(hitInfo.normal*glm::sign(dotn), state);
			float dt = glm::dot(nr, hitInfo.normal);
			f = material->t*0.5f;
			pdf = abs(dt)*0.5f;
			break;
		}
		//sample random direction in opposite hemicircle
		case SPECULARGLASS:{
			float rIndex = getRefractionIndex(rayInfo.wavelength / 1000.0f, material->b, material->c);
			float etaA = 1.0f;
			float etaB = rIndex;
			float F = frDielectric(-dotn, etaA, etaB);
			float c = rand1f(state);
			if (c < F){
				nr = glm::reflect(ray.rd, hitInfo.normal);
				float dt = glm::dot(nr, hitInfo.normal);
				pdf = F;
				f = F * material->r / glm::abs(dt);
				//f *= 0.0f;
			}
			else{
				bool entering = dotn < 0;
				float etaI = entering ? etaA : etaB;
				float etaT = entering ? etaB : etaA;
				nr = glm::refract(ray.rd, -hitInfo.normal*glm::sign(dotn), etaI / etaT);
				float dt = glm::dot(nr, hitInfo.normal);
				glm::vec3 ft = material->t * (1.0f - F);
				pdf = 1.0f - F;
				if ((nr.x*nr.x + nr.y*nr.y) < 0.01f)
					f = glm::vec3(0.0f);
				else
					f = ft / glm::abs(dt);
				rayInfo.speed = rayInfo.speed*(etaT / etaI);
			}
			break;
		}
		case SPECULARREFL:{
			float rIndex = getRefractionIndex(rayInfo.wavelength / 1000.0f, material->b, material->c);
			if (dotn >= 0.0f){
				rayInfo.depth = 0;
				break;
			}
			nr = glm::reflect(ray.rd, hitInfo.normal);
			float dt = glm::dot(nr, hitInfo.normal);
			f = material->r*frDielectric(-dotn, 1.0, rIndex) / abs(dt);
			pdf = 1.0;
			break;
		}
	}
	lr = rayInfo.color * f * glm::abs(glm::dot(nr, hitInfo.normal)) / pdf;
	rayInfo.ro = hitInfo.point + hitInfo.normal*glm::sign(glm::dot(nr, hitInfo.normal))*0.0005f;//new position
	rayInfo.rd = nr;
	rayInfo.color = lr;
}

__global__ void kernelLinePulse(RayInfo* infoArray, VBOInfo *vbo, const CudaScene scene, hiprandState_t*states, int* rayCount, int size, float time, float dt){
	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index >= size){
		return;
	}
	hiprandState_t * state = &states[index];
	/*atomicAdd(rayCount, 1);
	vbo[index * 2] = VBOInfo{ glm::vec2(index),glm::vec2(0.0f), glm::vec3(1.0f), 0.1 };
	vbo[index * 2 + 1] = VBOInfo{ glm::vec2(index), glm::vec2(0.0f), glm::vec3(0.5f, 1.0f, 0.1f), 0.4 };
	infoArray[index].objectIndex = index;
	return;*/
	RayInfo rayInfo = infoArray[index];
	Ray2D ray;
	float currentTime = rayInfo.time;
	if (rayInfo.depth == 0){
		atomicAdd(rayCount, 1);//compute total number of rays for color normalization
		ray = genRay(scene.lights, scene.lightCount, rayInfo, currentTime, state);
	}
	else{
		ray.ro = rayInfo.ro;
		ray.rd = rayInfo.rd;
	}
	float angle = glm::atan(ray.rd.y, ray.rd.x);
	glm::vec3 currentColor = rayInfo.color;
	HitInfo hitInfo;
	if (findIntersection(ray, scene.objects, scene.objCount, scene.lights, scene.lightCount, hitInfo)){
		rayInfo.time = currentTime + glm::distance(ray.ro, hitInfo.point) / rayInfo.speed;
		if (hitInfo.material == -1){
			rayInfo.depth = 0;
			rayInfo.ro = hitInfo.point;
		}
		else{
			shade(ray, hitInfo, scene.materials, rayInfo, state);
		}
	}
	else{
		glm::vec2 t2 = traceRect(ray, glm::vec2(-1.0f), glm::vec2(1.0f));
		float t = glm::max(t2.y, 0.0f);
		//intersect ray with camera bounds
		//set t value

		rayInfo.ro = ray.ro + ray.rd*t;
		rayInfo.depth = 0;
		rayInfo.time = currentTime + t / rayInfo.speed;
	}
	if (rayInfo.depth>scene.maxDepth)
		rayInfo.depth = 0;
	//russian rullete path termination
	else if (rayInfo.depth > scene.minDepth){
		float q = glm::max((float)0.05f, 1.0f - glm::max(rayInfo.color.x, glm::max(rayInfo.color.y, rayInfo.color.z)));
		if (rand1f(state) < q)
			rayInfo.depth = 0;
		else
			rayInfo.color /= 1 - q;
	}
	float t2 = time + dt;
	if (t2 <= rayInfo.time)
		rayInfo.depth = 0;
	if (time >= rayInfo.time){
		vbo[index * 2] = VBOInfo{ rayInfo.ro, glm::vec2(0.0f), glm::vec3(0.0f), angle };
		vbo[index * 2 + 1] = VBOInfo{ rayInfo.ro, glm::vec2(0.0f), glm::vec3(0.0f), angle };
	}
	else if (t2 <= currentTime){
		vbo[index * 2] = VBOInfo{ rayInfo.ro, glm::vec2(0.0f), glm::vec3(0.0f), angle };
		vbo[index * 2 + 1] = VBOInfo{ rayInfo.ro, glm::vec2(0.0f), glm::vec3(0.0f), angle };
	}
	else{
		float t1 = glm::max(time, currentTime); 
		t2 = glm::min(t2, rayInfo.time);
		vbo[index * 2] = VBOInfo{ glm::mix(ray.ro, rayInfo.ro, (t1 - currentTime) / (rayInfo.time - currentTime)), glm::vec2(0.0f), currentColor, angle };
		vbo[index * 2 + 1] = VBOInfo{ glm::mix(ray.ro, rayInfo.ro, (t2 - currentTime) / (rayInfo.time - currentTime)), glm::vec2(0.0f), currentColor, angle };
	}
	//put vboinfo with old ray info
	infoArray[index] = rayInfo;
}

__global__ void kernelLine(RayInfo* infoArray, VBOInfo *vbo, const CudaScene scene, hiprandState_t*states, int* rayCount, int size, float time){
	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index >= size){
		return;
	}
	hiprandState_t * state = &states[index];
	RayInfo rayInfo = infoArray[index];
	Ray2D ray;
	float currentTime = rayInfo.time;
	if (rayInfo.depth == 0){
		atomicAdd(rayCount, 1);//compute total number of rays for color normalization
		ray=genRay(scene.lights,scene.lightCount,rayInfo,currentTime,state);
	}
	else{
		ray.ro = rayInfo.ro;
		ray.rd = rayInfo.rd;
	}
	float angle = glm::atan(ray.rd.y, ray.rd.x);
	glm::vec3 currentColor = rayInfo.color;
	HitInfo hitInfo;
	if (findIntersection(ray, scene.objects, scene.objCount, scene.lights, scene.lightCount, hitInfo)){
		rayInfo.time = currentTime + glm::distance(ray.ro, hitInfo.point) / rayInfo.speed;
		if (hitInfo.material == -1){
			rayInfo.depth = 0;//gen new ray next time
			rayInfo.ro = hitInfo.point;
		}
		else{
			shade(ray,hitInfo,scene.materials,rayInfo,state);
		}
	}
	else{
		glm::vec2 t2 = traceRect(ray, glm::vec2(-1.0f), glm::vec2(1.0f));
		float t = glm::max(t2.y, 0.0f);
		//intersect ray with camera bounds
		//set t value
		rayInfo.ro = ray.ro + ray.rd*t;
		rayInfo.depth = 0;
		rayInfo.time = currentTime + t / rayInfo.speed;
	}
	if (rayInfo.depth>scene.maxDepth)
		rayInfo.depth = 0;
	//russian rullete path termination
	else if (rayInfo.depth > scene.minDepth){
		float q = glm::max((float)0.05f, 1.0f - glm::max(rayInfo.color.x, glm::max(rayInfo.color.y, rayInfo.color.z)));
		if (rand1f(state) < q)
			rayInfo.depth = 0;
		else
			rayInfo.color /= 1 - q;
	}

	if (rayInfo.time >= time&& currentTime <= time){
		vbo[index * 2] = VBOInfo{ ray.ro, glm::vec2(0.0f), currentColor, angle };
		vbo[index * 2 + 1] = VBOInfo{ glm::mix(ray.ro, rayInfo.ro, (time - currentTime) / (rayInfo.time - currentTime)), glm::vec2(0.0f), currentColor, angle };
		rayInfo.depth = 0;
	}
	else{
		vbo[index * 2] = VBOInfo{ ray.ro, glm::vec2(0.0f), currentColor, angle };
		vbo[index * 2 + 1] = VBOInfo{ rayInfo.ro, glm::vec2(0.0f), currentColor, angle };
	}
	//put vboinfo with old ray info
	infoArray[index] = rayInfo;
}

__global__ void curandInitKernel(hiprandState_t*states, int seed, int size){
	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;
	if (index >= size){
		return;
	}
	hiprandState_t * state = &states[index];
	hiprand_init(seed, index, 0, state);
}